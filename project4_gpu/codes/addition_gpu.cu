//
// include files
//


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

//
// kernel routine
// 

__global__ void VecAdd(float* A, float* B, float* C)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    C[i] = A[i] + B[i];
}


//
// main code
//

int main(int argc, char **argv)
{
    hipSetDevice(1);
    // Input the vector length
    int N = atoi(argv[1]);

    // Number of bytes to allocate for N float
    size_t bytes = N*sizeof(float);

    // Generate randomly vectors A and B
    float *A = (float *)malloc(bytes);
    float *B = (float *)malloc(bytes);
    float *C = (float *)malloc(bytes);
    
    // Allocate memory for arrays d_A, d_B, and d_C on device
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    for (int i = 0; i < N; i++)
    {
        A[i] = rand()%100;
        B[i] = rand()%100;
    }

    hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice);

    // Kernel invocation
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C);

    // Copy data from device array d_C to host array C
    hipMemcpy(C, d_C, bytes, hipMemcpyDeviceToHost);

    int s = 0;
    for (int j = 0; j < N; j++) s += C[j];
        
    printf("\nGPU Vector Length: %d Sum: %d\n", N, s);

    // Free CPU memory
    free(A);
    free(B);
    free(C);

    // Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // CUDA exit -- needed to flush printf write buffer
    hipDeviceReset();

    return 1;
}